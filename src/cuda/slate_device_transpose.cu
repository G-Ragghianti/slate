#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// Copyright (c) 2017, University of Tennessee
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the University of Tennessee nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL UNIVERSITY OF TENNESSEE BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//------------------------------------------------------------------------------
// This research was supported by the Exascale Computing Project (17-SC-20-SC),
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.
//------------------------------------------------------------------------------
// For assistance with SLATE, email <slate-user@icl.utk.edu>.
// You can also join the "SLATE User" Google group by going to
// https://groups.google.com/a/icl.utk.edu/forum/#!forum/slate-user,
// signing in with your Google credentials, and then clicking "Join group".
//------------------------------------------------------------------------------

#include "slate/internal/device.hh"
#include "slate_device_util.cuh"

#include <cstdio>
#include <hip/hip_complex.h>

namespace slate {
namespace device {

// internal blocking
// 16 x 16 thread block = 256 threads
// 32 x 32 thread block = 1024 threads
static const int ib = 16;

//------------------------------------------------------------------------------
/// Device routine handles one matrix.
/// Thread block grid:
/// x = batch index (ignored here; see batch kernel),
/// y = block row index,
/// z = block col index.
/// Each thread block is ib-by-ib threads and does one ib-by-ib block of an
/// n-by-n matrix.
///
/// Let nt = ceildiv( n, ib ) be the number of blocks for one n-by-n matrix.
/// An even number of blocks uses an (nt + 1) by (nt/2) grid.
/// Example: for nt = 4 blocks, y by z = 5 by 2 grid:
///   [ A00  A01 ]
///   [----. A11 ]                  [ A10  .  |  .   .  ]
///   [ A10 '----]                  [ A20 A21 |  .   .  ]
///   [ A20  A21 ] covers matrix as [ A30 A31 | A00  .  ]
///   [ A30  A31 ]                  [ A40 A41 | A01 A11 ]
///   [ A40  A41 ]
///
/// An odd number of blocks uses an (nt) by (nt + 1)/2 grid.
/// Example: for nt = 5 blocks, y by z = 5 by 3 grid:
///   [ A00 | A01   A02 ]
///   [     '----.      ]                  [ A00  .   .  |  .   .  ]
///   [ A10   A11 | A12 ]                  [ A10 A11  .  |  .   .  ]
///   [           '-----] covers matrix as [ A20 A21 A22 |  .   .  ]
///   [ A20   A21   A22 ]                  [ A30 A31 A32 | A01  .  ]
///   [ A30   A31   A32 ]                  [ A40 A41 A42 | A02 A12 ]
///   [ A40   A41   A42 ]
///
template <typename scalar_t>
__device__ void transpose_func(
    int n,
    scalar_t* A, int64_t lda)
{
    // +1 to avoid memory bank conflicts.
    __shared__ scalar_t sA1[ ib ][ ib+1 ];
    __shared__ scalar_t sA2[ ib ][ ib+1 ];

    // i, j are row & column indices of top-left corner of each block.
    // ii, jj are row & column offsets within each block.
    int ii = threadIdx.x;
    int jj = threadIdx.y;

    int i, j;
    if (gridDim.y - 1 == gridDim.z*2) {
        // Even number of blocks.
        //assert( ceildiv(n, ib) % 2 == 0 );
        bool lower = (blockIdx.y > blockIdx.z);
        i = (lower ? (blockIdx.y - 1) : (blockIdx.z + gridDim.z));
        j = (lower ? (blockIdx.z    ) : (blockIdx.y + gridDim.z));
    }
    else {
        // Odd number of blocks.
        //assert( ceildiv(n, ib) % 2 == 1 );
        bool lower = (blockIdx.y >= blockIdx.z);
        i = (lower ? blockIdx.y : (blockIdx.z + gridDim.z - 1));
        j = (lower ? blockIdx.z : (blockIdx.y + gridDim.z    ));
    }
    i *= ib;
    j *= ib;

    scalar_t* A1 = A + i + ii + (j + jj)*lda;  // A(i, j)
    if (i == j) { // diagonal block
        // Load block A(i, j) into shared memory sA1.
        if (i + ii < n  &&  j + jj < n) {
            sA1[jj][ii] = *A1;
        }
        __syncthreads();

        // Save transposed block, A(i, j) = trans(sA1).
        if (i + ii < n  &&  j + jj < n) {
            *A1 = sA1[ii][jj];
        }
    }
    else { // off-diagonal block
        scalar_t* A2 = A + j + ii + (i + jj)*lda;  // A(j, i)
        // Load blocks A(i, j) and A(j, i) into shared memory sA1 and sA2.
        if (i + ii < n  &&  j + jj < n) {
            sA1[jj][ii] = *A1;
        }
        if (j + ii < n  &&  i + jj < n) {
            sA2[jj][ii] = *A2;
        }
        __syncthreads();

        // Save transposed blocks, A(i, j) = trans(sA2), A(j, i) = trans(sA1).
        if (i + ii < n && j + jj < n) {
            *A1 = sA2[ii][jj];
        }
        if (j + ii < n && i + jj < n) {
            *A2 = sA1[ii][jj];
        }
    }
}

//------------------------------------------------------------------------------
template <typename scalar_t>
__global__ void transpose_kernel(
    int n,
    scalar_t* A, int64_t lda)
{
    transpose_func(n, A, lda);
}

//------------------------------------------------------------------------------
template <typename scalar_t>
__global__ void transpose_batch_kernel(
    int n,
    scalar_t** Aarray, int64_t lda)
{
    transpose_func(n, Aarray[blockIdx.x], lda);
}

//------------------------------------------------------------------------------
/// Physically transpose a square matrix in place.
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 0.
///
/// @param[in,out] A
///     A square n-by-n matrix stored in an lda-by-n array in GPU memory.
///     On output, A is transposed.
///
/// @param[in] lda
///     Leading dimension of A. lda >= n.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t>
void transpose(
    int64_t n,
    scalar_t* A, int64_t lda,
    hipStream_t stream)
{
    if (n <= 1)
        return;
    assert(lda >= n);

    int nt = ceildiv( n, int64_t(ib) );
    assert(nt <= 65535);                // CUDA limitation

    // Need 1/2 * (nt + 1) * nt to cover lower triangle and diagonal of matrix.
    // Block assignment differs depending on whether nt is odd or even.
    dim3 blocks;
    if (nt % 2 == 0) {
        // even blocks
        blocks = { 1, uint(nt + 1), uint(nt/2) };
    }
    else {
        // odd blocks
        blocks = { 1, uint(nt), uint((nt + 1)/2) };
    }
    dim3 threads( ib, ib );

    transpose_kernel<<< blocks, threads, 0, stream >>>
        (n, A, lda);

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
/// Physically transpose a batch of square matrices in place.
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 0.
///
/// @param[in,out] Aarray
///     Array in GPU memory of dimension batch_count, containing pointers to
///     matrices, where each Aarray[k] is a square n-by-n matrix stored in an
///     lda-by-n array in GPU memory.
///     On output, each Aarray[k] is transposed.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= n.
///
/// @param[in] batch_count
///     Size of Aarray. batch_count >= 0.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t>
void transpose_batch(
    int64_t n,
    scalar_t** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream)
{
    if (batch_count < 0 || n <= 1)
        return;
    assert(lda >= n);

    int nt = ceildiv( n, int64_t(ib) );
    assert(nt <= 65535);                // CUDA limitation
    assert(batch_count <= 2147483647);  // CUDA limitation, 2^31 - 1

    // Need 1/2 * (nt + 1) * nt to cover lower triangle and diagonal of matrix.
    // Block assignment differs depending on whether nt is odd or even.
    dim3 blocks;
    if (nt % 2 == 0) {
        // even blocks
        blocks = { uint(batch_count), uint(nt + 1), uint(nt/2) };
    }
    else {
        // odd blocks
        blocks = { uint(batch_count), uint(nt), uint((nt + 1)/2) };
    }
    dim3 threads( ib, ib );

    transpose_batch_kernel<<< blocks, threads, 0, stream >>>
        (n, Aarray, lda);

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
// Explicit instantiations.
template
void transpose(
    int64_t n,
    float* A, int64_t lda,
    hipStream_t stream);

template
void transpose(
    int64_t n,
    double* A, int64_t lda,
    hipStream_t stream);

template
void transpose(
    int64_t n,
    hipFloatComplex* A, int64_t lda,
    hipStream_t stream);

template
void transpose(
    int64_t n,
    hipDoubleComplex* A, int64_t lda,
    hipStream_t stream);

// ----------------------------------------
template
void transpose_batch(
    int64_t n,
    float** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);

template
void transpose_batch(
    int64_t n,
    double** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);

template
void transpose_batch(
    int64_t n,
    hipFloatComplex** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);

template
void transpose_batch(
    int64_t n,
    hipDoubleComplex** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);

} // namespace device
} // namespace slate
