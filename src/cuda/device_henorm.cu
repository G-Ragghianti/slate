#include "hip/hip_runtime.h"
// Copyright (c) 2017-2020, University of Tennessee. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause
// This program is free software: you can redistribute it and/or modify it under
// the terms of the BSD 3-Clause license. See the accompanying LICENSE file.

#include "slate/Exception.hh"
#include "slate/internal/device.hh"

#include "device_util.cuh"

#include <cstdio>

namespace slate {
namespace device {

//------------------------------------------------------------------------------
__host__ __device__
inline
float real(float a)
{
    return a;
}

__host__ __device__
float imag(float a)
{
    return 0;
}

//------------------------------------------------------------------------------
__host__ __device__
inline
double real(double a)
{
    return a;
}

__host__ __device__
inline
double imag(double a)
{
    return 0;
}

//------------------------------------------------------------------------------
__host__ __device__
inline
float real(hipFloatComplex a)
{
    return a.x;
}

__host__ __device__
inline
float imag(hipFloatComplex a)
{
    return a.y;
}

//------------------------------------------------------------------------------
__host__ __device__
inline
double real(hipDoubleComplex a)
{
    return a.x;
}

__host__ __device__
inline
double imag(hipDoubleComplex a)
{
    return a.y;
}

//------------------------------------------------------------------------------
/// Finds the largest absolute value of elements, for each tile in tiles.
/// Each thread block deals with one tile.
/// Each thread deals with one row, followed by a reduction.
/// Uses dynamic shared memory array of length sizeof(real_t) * n.
/// Kernel assumes non-trivial tiles (n >= 1).
/// Launched by henorm().
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 1.
///     Also the number of threads per block (blockDim.x), hence,
///
/// @param[in] tiles
///     Array of tiles of dimension gridDim.x,
///     where each tiles[k] is an n-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= n.
///
/// @param[out] tiles_maxima
///     Array of dimension gridDim.x.
///     On exit, tiles_maxima[k] = max_{i, j} abs( A^(k)_(i, j) )
///     for tile A^(k).
///
template <typename scalar_t>
__global__ void henormMaxKernel(
    lapack::Uplo uplo,
    int64_t n,
    scalar_t const* const* tiles, int64_t lda,
    blas::real_type<scalar_t>* tiles_maxima)
{
    using real_t = blas::real_type<scalar_t>;
    scalar_t const* tile = tiles[blockIdx.x];
    int chunk;

    // Save partial results in shared memory.
    extern __shared__ char dynamic_data[];
    real_t* row_max = (real_t*) dynamic_data;

    // Each thread finds max of one row.
    // This does coalesced reads of one column at a time in parallel.
    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        chunk = idx % blockDim.x;

        scalar_t const* row = &tile[idx];
        if (idx < blockDim.x) {
            row_max[chunk] = 0;
        }

        real_t max = 0;
        if (uplo == lapack::Uplo::Lower) {
            for (int64_t j = 0; j < idx && j < n; ++j) // strictly lower
                max = max_nan(max, abs(row[j*lda]));
            int64_t j = idx;
            max = max_nan(max, abs( real( row[j*lda] )));  // diag (real)
        }
        else {
            // Loop backwards (n-1 down to i) to maintain coalesced reads.
            for (int64_t j = n-1; j > idx; --j) // strictly upper
                max = max_nan(max, abs(row[j*lda]));
            int64_t j = idx;
            max = max_nan(max, abs( real( row[j*lda] )));  // diag (real)
        }
        row_max[chunk] = max_nan(max, row_max[chunk]);
    }

    // Reduction to find max of tile.
    __syncthreads();
    max_nan_reduce(blockDim.x, threadIdx.x, row_max);
    if (threadIdx.x == 0) {
        tiles_maxima[blockIdx.x] = row_max[0];
    }
}

//------------------------------------------------------------------------------
/// Sum of absolute values of each column of elements, for each tile in tiles.
/// Each thread block deals with one tile.
/// Each thread deals with one column.
/// Kernel assumes non-trivial tiles (n >= 1).
/// Launched by henorm().
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 1.
///     Also the number of threads per block (blockDim.x), hence,
///
/// @param[in] tiles
///     Array of tiles of dimension gridDim.x,
///     where each tiles[k] is an n-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= n.
///
/// @param[out] tiles_sums
///     Array of dimension gridDim.x * ldv.
///     On exit, tiles_sums[k*ldv + j] = max_{i} abs( A^(k)_(i, j) )
///     for row j of tile A^(k).
///
/// @param[in] ldv
///     Leading dimension of tiles_sums (values) array.
///
template <typename scalar_t>
__global__ void henormOneKernel(
    lapack::Uplo uplo,
    int64_t n,
    scalar_t const* const* tiles, int64_t lda,
    blas::real_type<scalar_t>* tiles_sums, int64_t ldv)
{
    using real_t = blas::real_type<scalar_t>;
    scalar_t const* tile = tiles[blockIdx.x];

    // Each thread sums one row/column.
    // todo: the row reads are coalesced, but the col reads are not coalesced
    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {

        scalar_t const* row    = &tile[idx];
        scalar_t const* column = &tile[lda*idx];
        real_t sum = 0;

        if (uplo == lapack::Uplo::Lower) {
            for (int64_t j = 0; j < idx; ++j) // strictly lower
                sum += abs(row[j*lda]);
            int64_t j = idx;
            sum += abs( real( row[j*lda] )); // diag (real)
            for (int64_t i = idx + 1; i < n; ++i) // strictly lower
                sum += abs(column[i]);
        }
        else {
            // Loop backwards (n-1 down to i) to maintain coalesced reads.
            for (int64_t j = n-1; j > idx; --j) // strictly upper
                sum += abs(row[j*lda]);
            int64_t j = idx;
            sum += abs( real( row[j*lda] )); // diag (real)
            for (int64_t i = 0; i < idx && i < n; ++i) // strictly upper
                sum += abs(column[i]);
        }
        tiles_sums[blockIdx.x*ldv + idx] = sum;
    }
}

//------------------------------------------------------------------------------
/// Sum of squares, in scaled representation, for each tile in tiles.
/// Each thread block deals with one tile.
/// Each thread deals with one row, followed by a reduction.
/// Kernel assumes non-trivial tiles (n >= 1).
/// Launched by henorm().
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 1.
///     Also the number of threads per block, hence,
///
/// @param[in] tiles
///     Array of tiles of dimension blockDim.x,
///     where each tiles[k] is an n-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= n.
///
/// @param[out] tiles_values
///     Array of dimension 2 * blockDim.x.
///     On exit,
///         tiles_values[2*k + 0] = scale
///         tiles_values[2*k + 1] = sumsq
///     such that scale^2 * sumsq = sum_{i,j} abs( A^(k)_{i,j} )^2
///     for tile A^(k).
///
template <typename scalar_t>
__global__ void henormFroKernel(
    lapack::Uplo uplo,
    int64_t n,
    scalar_t const* const* tiles, int64_t lda,
    blas::real_type<scalar_t>* tiles_values)
{
    using real_t = blas::real_type<scalar_t>;
    scalar_t const* tile = tiles[blockIdx.x];
    int chunk;

    // Save partial results in shared memory.
    extern __shared__ char dynamic_data[];
    real_t* row_scale = (real_t*) &dynamic_data[0];
    real_t* row_sumsq = &row_scale[blockDim.x];

    // Each thread finds sum-of-squares of one row.
    // This does coalesced reads of one column at a time in parallel.
    for (int idx = threadIdx.x; idx < n; idx += blockDim.x) {
        real_t scale = 0;
        real_t sumsq = 1;
        chunk = idx % blockDim.x;
        scalar_t const* row = &tile[idx];

        if (uplo == lapack::Uplo::Lower) {
            for (int64_t j = 0; j < idx && j < n; ++j) // strictly lower
                add_sumsq(scale, sumsq, abs(row[j*lda]));
            // double for symmetric entries
            sumsq *= 2;
            // diagonal (real)
            add_sumsq(scale, sumsq, abs( real( row[idx*lda] )));
        }
        else {
            // Loop backwards (n-1 down to i) to maintain coalesced reads.
            for (int64_t j = n-1; j > idx; --j) // strictly upper
                add_sumsq(scale, sumsq, abs(row[j*lda]));
            // double for symmetric entries
            sumsq *= 2;
            // diagonal (real)
            add_sumsq(scale, sumsq, abs( real( row[idx*lda] )));
        }

        if (idx < blockDim.x) {
            row_scale[chunk] = 0;
            row_sumsq[chunk] = 1;
        }
        combine_sumsq(row_scale[chunk], row_sumsq[chunk], scale, sumsq);
        __syncthreads();
    }

    // Reduction to find sum-of-squares of tile.
    // todo: parallel reduction.
    if (threadIdx.x == 0) {
        real_t tile_scale = row_scale[0];
        real_t tile_sumsq = row_sumsq[0];
        for (int64_t chunk = 1; chunk < blockDim.x && chunk < n; ++chunk) {
            combine_sumsq(tile_scale, tile_sumsq, row_scale[chunk], row_sumsq[chunk]);
        }

        tiles_values[blockIdx.x*2 + 0] = tile_scale;
        tiles_values[blockIdx.x*2 + 1] = tile_sumsq;
    }
}

//------------------------------------------------------------------------------
/// Batched routine that returns the largest absolute value of elements for
/// each tile in Aarray. Sets
///     tiles_maxima[k] = max_{i, j}( abs( A^(k)_(i, j) )),
/// for each tile A^(k), where
/// A^(k) = Aarray[k],
/// k = 0, ..., blockDim.x-1,
/// i = 0, ..., n-1,
/// j = 0, ..., n-1.
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 0.
///
/// @param[in] Aarray
///     Array in GPU memory of dimension batch_count, containing pointers to tiles,
///     where each Aarray[k] is an n-by-n matrix stored in an lda-by-n array in GPU memory.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= n.
///
/// @param[out] values
///     Array in GPU memory, dimension batch_count * ldv.
///     - Norm::Max: ldv = 1.
///         On exit, values[k] = max_{i, j} abs( A^(k)_(i, j) )
///         for 0 <= k < batch_count.
///
///     - Norm::One: ldv >= n.
///         On exit, values[k*ldv + j] = sum_{i} abs( A^(k)_(i, j) )
///         for 0 <= k < batch_count, 0 <= j < n.
///
///     - Norm::Inf: for symmetric, same as Norm::One
///
///     - Norm::Max: ldv = 2.
///         On exit,
///             values[k*2 + 0] = scale_k
///             values[k*2 + 1] = sumsq_k
///         where scale_k^2 sumsq_k = sum_{i,j} abs( A^(k)_(i, j) )^2
///         for 0 <= k < batch_count.
///
/// @param[in] ldv
///     Leading dimension of tiles_sums (values) array.
///
/// @param[in] batch_count
///     Size of Aarray. batch_count >= 0.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t>
void henorm(
    lapack::Norm norm, lapack::Uplo uplo,
    int64_t n,
    scalar_t const* const* Aarray, int64_t lda,
    blas::real_type<scalar_t>* values, int64_t ldv, int64_t batch_count,
    blas::Queue &queue)
{
    using real_t = blas::real_type<scalar_t>;
    int64_t nb = 512;

    // quick return
    if (batch_count == 0)
        return;

    //---------
    // max norm
    if (norm == lapack::Norm::Max) {
        if (n == 0) {
            blas::device_memset(values, 0, batch_count, queue);
        }
        else {
            assert(ldv == 1);
            henormMaxKernel<<<batch_count, nb, sizeof(real_t) * nb, queue.stream()>>>
                (uplo, n, Aarray, lda, values);
        }
    }
    //---------
    // one norm
    else if (norm == lapack::Norm::One || norm == lapack::Norm::Inf) {
        if (n == 0) {
            blas::device_memset(values, 0, batch_count * n, queue);
        }
        else {
            assert(ldv >= n);
            henormOneKernel<<<batch_count, nb, 0, queue.stream()>>>
                (uplo, n, Aarray, lda, values, ldv);
        }
    }
    //---------
    // Frobenius norm
    else if (norm == lapack::Norm::Fro) {
        if (n == 0) {
            blas::device_memset(values, 0, batch_count * 2, queue);
        }
        else {
            assert(ldv == 2);
            henormFroKernel<<<batch_count, nb, sizeof(real_t) * nb * 2, queue.stream()>>>
                (uplo, n, Aarray, lda, values);
        }
    }

    hipError_t error = hipGetLastError();
    slate_assert(error == hipSuccess);
}

//------------------------------------------------------------------------------
// Explicit instantiations.
template
void henorm(
    lapack::Norm norm, lapack::Uplo uplo,
    int64_t n,
    float const* const* Aarray, int64_t lda,
    float* values, int64_t ldv, int64_t batch_count,
    blas::Queue &queue);

template
void henorm(
    lapack::Norm norm, lapack::Uplo uplo,
    int64_t n,
    double const* const* Aarray, int64_t lda,
    double* values, int64_t ldv, int64_t batch_count,
    blas::Queue &queue);

template
void henorm(
    lapack::Norm norm, lapack::Uplo uplo,
    int64_t n,
    hipFloatComplex const* const* Aarray, int64_t lda,
    float* values, int64_t ldv, int64_t batch_count,
    blas::Queue &queue);

template
void henorm(
    lapack::Norm norm, lapack::Uplo uplo,
    int64_t n,
    hipDoubleComplex const* const* Aarray, int64_t lda,
    double* values, int64_t ldv, int64_t batch_count,
    blas::Queue &queue);

} // namespace device
} // namespace slate
